#include "hip/hip_runtime.h"
// #define __INPUTDOT__
#include <usv_dynamics.cuh>
#define MAX_CTRL 20.0f

// 定义符号函数
float sign(float x)
{
    return (x > 0) - (x < 0);
}

namespace heron
{
    // 构造函数
    USVDynamics::USVDynamics(hipStream_t stream)
        : Dynamics<USVDynamics, USVDynamicsParams>(stream)
    {
        this->params_ = USVDynamicsParams();
    }

    // 计算动力学方程
    void USVDynamics::computeDynamics(const Eigen::Ref<const state_array> &state,
                                      const Eigen::Ref<const control_array> &control,
                                      Eigen::Ref<state_array> state_der)
    {
        // Extract state variables
        float psi = state(2); // Heading Angle
        float u = state(3);   // Surge Velocity
        float v = state(4);   // Sway Velocity
        float r = state(5);   // Yaw Rate

        // Extract control inputs
        float S_left = control(0);  // Left Thruster Input
        float S_right = control(1); // Right Thruster Input
        Eigen::Matrix3f Jacobian = (Eigen::Matrix3f() << cosf(psi), -sinf(psi), 0, sinf(psi), cosf(psi), 0, 0, 0, 1).finished();
        Eigen::Vector3f nu = Jacobian * Eigen::Vector3f(u, v, r); // Transform the control inputs to the body frame
        // Compute the dynamics
        state_der(0) = nu(0);
        state_der(1) = nu(1);
        state_der(2) = nu(2);
        state_der(3) = u_dot(u, v, r, S_left, S_right);
        state_der(4) = v_dot(u, v, r, S_left, S_right);
        state_der(5) = r_dot(u, v, r, S_left, S_right);
    }

    // 连续动力学方程（CUDA设备）
    __device__ void USVDynamics::computeDynamics(float *state, float *control, float *state_der,
                                                 float *theta_s)
    {
        // Extract state variables
        float psi = state[2]; // Heading Angle
        float u = state[3];   // Surge Velocity
        float v = state[4];   // Sway Velocity
        float r = state[5];   // Yaw Rate

        // Extract control inputs
        float S_left = control[0];  // Left Thruster Input
        float S_right = control[1]; // Right Thruster Input

        // Compute the dynamics
        state_der[0] = __cosf(psi) * u - __sinf(psi) * v;
        state_der[1] = __sinf(psi) * u + __cosf(psi) * v;
        state_der[2] = r;
        state_der[3] = u_dot(u, v, r, S_left, S_right);
        state_der[4] = v_dot(u, v, r, S_left, S_right);
        state_der[5] = r_dot(u, v, r, S_left, S_right);
    }

    // 从输入数据映射到状态
    Dynamics<USVDynamics, USVDynamicsParams>::state_array
    USVDynamics::stateFromMap(const std::map<std::string, float> &map)
    {
        state_array s;
        s(0) = map.at("POS_X");
        s(1) = map.at("POS_Y");
        s(2) = map.at("POS_PSI");
        s(3) = map.at("VEL_U");
        s(4) = map.at("VEL_V");
        s(5) = map.at("VEL_R");
        return s;
    }

    void USVDynamics::printState(float *state)
    {
        printf("X position: %.2f; Y position: %.2f; Heading Angle: %.2f \n", state[0], state[1], state[2]);
    }

    void USVDynamics::printState(const float *state)
    {
        printf("X position: %.2f; Y position: %.2f; Heading Angle: %.2f \n", state[0], state[1], state[2]);
    }

    // 施加控制约束（主机）
    __host__ void USVDynamics::enforceConstraints(Eigen::Ref<state_array> state, Eigen::Ref<control_array> control)
    {
        // 声明 state 未使用
        (void)state;
        control = control.cwiseMin(MAX_CTRL).cwiseMax(-MAX_CTRL); // 限制控制量在 -MAX_CTRL 到 MAX_CTRL 之间
    }

    // 施加控制约束（CUDAs）
    __device__ void USVDynamics::enforceConstraints(float *state, float *control)
    {
        // TODO should control_rngs_ be a constant memory parameter
        int i, p_index, step;
        mppi::p1::getParallel1DIndex<mppi::p1::Parallel1Dir::THREAD_Y>(p_index, step);
        // parallelize setting the constraints with y dim
        for (i = p_index; i < CONTROL_DIM; i += step)
        {
            control[i] = fminf(fmaxf(-MAX_CTRL, control[i]), MAX_CTRL);
        }
    }
}
